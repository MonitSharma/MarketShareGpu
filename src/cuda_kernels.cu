#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"

#include "profiler.hpp"

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <math.h>
#include <iostream>

template <typename T>
T *copy_to_device(const std::vector<T> &host_vec, int64_t &n_bytes_alloc_total)
{
    T *device_ptr;
    const auto n_bytes_alloc = host_vec.size() * sizeof(T);
    hipMalloc(&device_ptr, n_bytes_alloc);
    hipMemcpy(device_ptr, host_vec.data(), n_bytes_alloc, hipMemcpyHostToDevice);

    n_bytes_alloc_total += n_bytes_alloc;
    return device_ptr;
}

GpuData::GpuData(const MarkShareFeas &ms_inst, const std::vector<size_t> &set1_scores, const std::vector<size_t> &set2_scores, const std::vector<size_t> &set3_scores, const std::vector<size_t> &set4_scores) : m_rows(ms_inst.m()), n_cols(ms_inst.n())
{
    this->matrix = copy_to_device(ms_inst.A(), n_bytes_alloc);
    this->rhs = copy_to_device(ms_inst.b(), n_bytes_alloc);

    this->set1_scores = copy_to_device(set1_scores, n_bytes_alloc);
    this->set2_scores = copy_to_device(set2_scores, n_bytes_alloc);
    this->set3_scores = copy_to_device(set3_scores, n_bytes_alloc);
    this->set4_scores = copy_to_device(set4_scores, n_bytes_alloc);
}

GpuData::~GpuData()
{
    hipFree(set1_scores);
    hipFree(set2_scores);
    hipFree(set3_scores);
    hipFree(set4_scores);

    hipFree(matrix);
    hipFree(rhs);

    hipFree(required_buffer);

    hipFree(search_buffer);
    hipFree(results_search_buffer);
}

template <typename T>
void GpuData::resize_buffer(T **buffer, size_t &buffer_size, size_t n_elems_required)
{
    if (n_elems_required > buffer_size)
    {
        size_t new_buffer_size = static_cast<size_t>(n_elems_required * 1.4 + 1);
        assert(new_buffer_size > n_elems_required);

        hipFree(*buffer);

        n_bytes_alloc += sizeof(T) * (new_buffer_size - buffer_size);
        hipMalloc(buffer, new_buffer_size * sizeof(T));

        buffer_size = new_buffer_size;
    }
}

void GpuData::copy_pairs_required(const std::vector<std::pair<size_t, size_t>> &pairs)
{
    size_t len_needed = pairs.size();

    resize_buffer(&required_buffer, len_required_buffer, len_needed);

    size_t *pairs_required = (size_t *)(required_buffer);
    hipMemcpy(pairs_required, pairs.data(), pairs.size() * 2 * sizeof(size_t), hipMemcpyHostToDevice);
    n_required = len_needed;
}

void GpuData::copy_pairs_search(const std::vector<std::pair<size_t, size_t>> &pairs)
{
    size_t len_needed = pairs.size();
    resize_buffer(&search_buffer, len_search_buffer, len_needed);
    resize_buffer(&results_search_buffer, len_results_buffer, len_needed);

    size_t *pairs_search = (size_t *)(search_buffer);
    hipMemcpy(pairs_search, pairs.data(), pairs.size() * 2 * sizeof(size_t), hipMemcpyHostToDevice);
    n_search = len_needed;
}

void GpuData::copy_tuples(const std::vector<PairsTuple> &tuples)
{
    size_t len_needed = tuples.size() * 4;
    resize_buffer(&tuples_buffer, len_tuples_buffer, len_needed);

    hipError_t err = hipMemcpy(tuples_buffer, tuples.data(), len_needed * sizeof(size_t), hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    n_tuples = tuples.size();
}

template <bool ENCODE_REQUIRED>
__global__ void combine_and_encode_kernel(const size_t *__restrict__ scores1, const size_t *__restrict__ scores2, const size_t *__restrict__ rhs, size_t *__restrict__ pairs, size_t n_pairs, size_t encode_start, size_t encode_end, size_t m_rows)
{
    const int i_pair = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr size_t BASE = 10000; /* Maximum value per vector element */

    if (i_pair >= n_pairs)
        return;

    __int128_t key = 0;
    const size_t idx1 = pairs[2 * i_pair];
    const size_t idx2 = pairs[2 * i_pair + 1];

    for (size_t i_row = encode_start; i_row < encode_end; ++i_row)
    {
        /* Compute the pair's score of this row and add it (encoded) to key. */
        size_t row_score = scores1[idx1 * m_rows + i_row] + scores2[idx2 * m_rows + i_row];

        if (ENCODE_REQUIRED)
            row_score = rhs[i_row] - row_score;

        /* FMA. */
        key = key * BASE + row_score;
    }

    /* Offload key to the original pair position. */
    *(__int128_t *)(pairs + 2 * i_pair) = key;
}

/* Converts tuples into pairs. */
__global__ void flatten_tuples(const size_t *tuples, size_t n_tuples, size_t *pairs)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n_tuples)
        return;

    const size_t *tuple = tuples + 4 * idx;
    size_t pair_first = tuple[0];
    size_t pair_second_beg = tuple[1];
    size_t pair_second_end = tuple[2] + pair_second_beg;
    size_t pairs_offset = tuple[3];

    for (size_t second = pair_second_beg; second < pair_second_end; ++second)
    {
        pairs[2 * pairs_offset] = pair_first;
        pairs[2 * pairs_offset + 1] = second;
        ++pairs_offset;
    }
}

void combine_and_encode_first_five_gpu(GpuData &gpu_data, const std::vector<std::pair<size_t, size_t>> &pairs1, const std::vector<std::pair<size_t, size_t>> &pairs2)
{
    const size_t m_rows = gpu_data.m_rows;
    const size_t n_p1 = pairs1.size();
    const size_t n_p2 = pairs2.size();

    /* The shorter array will be encoded as required and will be sorted. */
    const bool encode_first_as_required = (n_p1 < n_p2);

    auto profiler = std::make_unique<ScopedProfiler>("Eval GPU: combine + encode  ");

    gpu_data.copy_pairs_required(encode_first_as_required ? pairs1 : pairs2);
    gpu_data.copy_pairs_search(encode_first_as_required ? pairs2 : pairs1);

    /* Each pair is treated by one single thread. */
    constexpr int block_dim = 128;
    int n_blocks_1 = (n_p1 + block_dim - 1) / block_dim;
    int n_blocks_2 = (n_p2 + block_dim - 1) / block_dim;

    size_t encode_start = 0;
    size_t encode_end = std::min(size_t(5), m_rows);

    if (encode_first_as_required)
    {
        combine_and_encode_kernel<true><<<n_blocks_1, block_dim>>>(gpu_data.set1_scores, gpu_data.set2_scores, gpu_data.rhs, (size_t *)gpu_data.required_buffer, n_p1, encode_start, encode_end, m_rows);
        combine_and_encode_kernel<false><<<n_blocks_2, block_dim>>>(gpu_data.set3_scores, gpu_data.set4_scores, gpu_data.rhs, (size_t *)gpu_data.search_buffer, n_p2, encode_start, encode_end, m_rows);
    }
    else
    {
        combine_and_encode_kernel<false><<<n_blocks_1, block_dim>>>(gpu_data.set1_scores, gpu_data.set2_scores, gpu_data.rhs, (size_t *)gpu_data.search_buffer, n_p1, encode_start, encode_end, m_rows);
        combine_and_encode_kernel<true><<<n_blocks_2, block_dim>>>(gpu_data.set3_scores, gpu_data.set4_scores, gpu_data.rhs, (size_t *)gpu_data.required_buffer, n_p2, encode_start, encode_end, m_rows);
    }

    profiler.reset();
}

void combine_and_encode_tuples_gpu(GpuData &gpu_data, const std::vector<PairsTuple> &tuples1, const std::vector<PairsTuple> &tuples2, size_t n_pairs1, size_t n_pairs2)
{
    auto profiler = std::make_unique<ScopedProfiler>("Eval GPU: combine + encode  ");

    /* Each tuple is treated by one warp. */
    const size_t m_rows = gpu_data.m_rows;

    /* The shorter array will be encoded as required. */
    const bool encode_first_as_required = (n_pairs1 < n_pairs2);

    const auto &required = encode_first_as_required ? tuples1 : tuples2;
    const auto &search = encode_first_as_required ? tuples2 : tuples1;

    const size_t *required_set1_scores = encode_first_as_required ? gpu_data.set1_scores : gpu_data.set3_scores;
    const size_t *required_set2_scores = encode_first_as_required ? gpu_data.set2_scores : gpu_data.set4_scores;

    const size_t *search_set1_scores = encode_first_as_required ? gpu_data.set3_scores : gpu_data.set1_scores;
    const size_t *search_set2_scores = encode_first_as_required ? gpu_data.set4_scores : gpu_data.set2_scores;

    const auto n_tuples_required = required.size();
    const auto n_tuples_search = search.size();
    gpu_data.n_required = encode_first_as_required ? n_pairs1 : n_pairs2;
    gpu_data.n_search = encode_first_as_required ? n_pairs2 : n_pairs1;

    /* Reserve space for hashes. */
    gpu_data.resize_buffer(&gpu_data.required_buffer, gpu_data.len_required_buffer, gpu_data.n_required);
    gpu_data.resize_buffer(&gpu_data.search_buffer, gpu_data.len_search_buffer, gpu_data.n_search);
    gpu_data.resize_buffer(&gpu_data.results_search_buffer, gpu_data.len_results_buffer, gpu_data.n_search);

    const int n_threads = 256;
    /* Copy and flatten the tuples. */
    gpu_data.copy_tuples(required);
    int n_blocks = (n_tuples_required + n_threads - 1) / n_threads;
    assert(n_blocks > 0);
    flatten_tuples<<<n_blocks, n_threads>>>(gpu_data.tuples_buffer, n_tuples_required, (size_t *)gpu_data.required_buffer);

    gpu_data.copy_tuples(search);
    n_blocks = (n_tuples_search + n_threads - 1) / n_threads;
    assert(n_blocks > 0);
    flatten_tuples<<<n_blocks, n_threads>>>(gpu_data.tuples_buffer, n_tuples_search, (size_t *)gpu_data.search_buffer);

    int n_blocks_1 = (gpu_data.n_required + n_threads - 1) / n_threads;
    int n_blocks_2 = (gpu_data.n_search + n_threads - 1) / n_threads;
    assert(n_blocks_1 > 0);
    assert(n_blocks_2 > 0);

    combine_and_encode_kernel<true><<<n_blocks_1, n_threads>>>(required_set1_scores, required_set2_scores, gpu_data.rhs, (size_t *)gpu_data.required_buffer, gpu_data.n_required, 0, m_rows, m_rows);
    combine_and_encode_kernel<false><<<n_blocks_2, n_threads>>>(search_set1_scores, search_set2_scores, gpu_data.rhs, (size_t *)gpu_data.search_buffer, gpu_data.n_search, 0, m_rows, m_rows);

    profiler.reset();
}

void combine_and_encode_gpu(GpuData &gpu_data, const std::vector<std::pair<size_t, size_t>> &pairs1, const std::vector<std::pair<size_t, size_t>> &pairs2)
{
    const size_t m_rows = gpu_data.m_rows;
    const size_t n_p1 = pairs1.size();
    const size_t n_p2 = pairs2.size();

    /* The shorter array will be encoded as required and will be sorted. */
    const bool encode_first_as_required = (n_p1 < n_p2);

    auto profiler = std::make_unique<ScopedProfiler>("Eval GPU: combine + encode  ");

    gpu_data.copy_pairs_required(encode_first_as_required ? pairs1 : pairs2);
    gpu_data.copy_pairs_search(encode_first_as_required ? pairs2 : pairs1);

    /* Each pair is treated by one single thread. */
    constexpr int block_dim = 128;
    int n_blocks_1 = (n_p1 + block_dim - 1) / block_dim;
    int n_blocks_2 = (n_p2 + block_dim - 1) / block_dim;

    if (encode_first_as_required)
    {
        combine_and_encode_kernel<true><<<n_blocks_1, block_dim>>>(gpu_data.set1_scores, gpu_data.set2_scores, gpu_data.rhs, (size_t *)gpu_data.required_buffer, n_p1, 0, m_rows, m_rows);
        combine_and_encode_kernel<false><<<n_blocks_2, block_dim>>>(gpu_data.set3_scores, gpu_data.set4_scores, gpu_data.rhs, (size_t *)gpu_data.search_buffer, n_p2, 0, m_rows, m_rows);
    }
    else
    {
        combine_and_encode_kernel<false><<<n_blocks_1, block_dim>>>(gpu_data.set1_scores, gpu_data.set2_scores, gpu_data.rhs, (size_t *)gpu_data.search_buffer, n_p1, 0, m_rows, m_rows);
        combine_and_encode_kernel<true><<<n_blocks_2, block_dim>>>(gpu_data.set3_scores, gpu_data.set4_scores, gpu_data.rhs, (size_t *)gpu_data.required_buffer, n_p2, 0, m_rows, m_rows);
    }

    profiler.reset();
}

std::pair<bool, __int128_t> find_equal_hash(GpuData &gpu_data)
{
    /* The shorter array will be encoded as required and will be sorted. */
    const size_t n_required = gpu_data.n_required;
    const size_t n_search = gpu_data.n_search;

    __int128_t *required = gpu_data.required_buffer;
    __int128_t *search = gpu_data.search_buffer;
    bool *result = gpu_data.results_search_buffer;

    /* Compute hashes of required vectors. */
    auto profiler = std::make_unique<ScopedProfiler>("Eval GPU: sort required     ");

    /* Sort the array of required keys. */
    thrust::sort(thrust::device, required, required + n_required);

    profiler = std::make_unique<ScopedProfiler>("Eval GPU: binary search     ");

    thrust::binary_search(thrust::device, required, required + n_required, search, search + n_search, result);

    profiler = std::make_unique<ScopedProfiler>("Eval GPU: check results     ");

    thrust::device_ptr<bool> result_ptr(result);
    auto iter = thrust::find(thrust::device, result_ptr, result_ptr + n_search, true);

    if (iter != result_ptr + n_search)
    {
        /* Get the position of the found element and copy back its (unsorted) search value. */
        size_t i_search = thrust::distance(result_ptr, iter);

        __int128_t val = 0;
        hipMemcpy(&val, search + i_search, sizeof(__int128_t), hipMemcpyDeviceToHost);

        return {true, val};
    }
    profiler.reset();

    return {false, 0};
}

std::pair<size_t, size_t> find_hash_positions_gpu(GpuData &gpu_data, __int128_t hash, size_t n_p1, size_t n_p2)
{
    const bool encode_first_as_required = (n_p1 < n_p2);

    __int128_t *required = gpu_data.required_buffer;
    __int128_t *search = gpu_data.search_buffer;

    const size_t n_required = gpu_data.n_required;
    const size_t n_search = gpu_data.n_search;

    auto iter_req = thrust::find(thrust::device, required, required + n_required, hash);
    auto iter_search = thrust::find(thrust::device, search, search + n_search, hash);

    assert(iter_req != required + n_required);
    assert(iter_search != search + n_search);

    auto pos_req = thrust::distance(required, iter_req);
    auto pos_search = thrust::distance(search, iter_search);

    if (encode_first_as_required)
        return {pos_req, pos_search};
    else
        return {pos_search, pos_req};
}
