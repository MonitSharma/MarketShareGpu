#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"

#include "profiler.hpp"

#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <iostream>

template <typename T>
T *copy_to_device(const std::vector<T> &host_vec)
{
    T *device_ptr;
    hipMalloc(&device_ptr, host_vec.size() * sizeof(T));
    hipMemcpy(device_ptr, host_vec.data(), host_vec.size() * sizeof(T), hipMemcpyHostToDevice);
    return device_ptr;
}

static void copy_subsets(const std::vector<std::vector<size_t>>& set_subsets, size_t** set_beg_gpu, size_t** sets_gpu)
{
    size_t subsets_size_total = 0;
    std::vector<size_t> set_beg;
    set_beg.reserve(set_subsets.size() + 1);

    for (const auto & subset : set_subsets)
    {
        set_beg.push_back(subsets_size_total);
        subsets_size_total += subset.size();
    }
    set_beg.push_back(subsets_size_total);

    std::vector<size_t> sets_flattened;
    sets_flattened.reserve(subsets_size_total);

    for (const auto& subset : set_subsets)
    {
        sets_flattened.insert(sets_flattened.end(), subset.begin(), subset.end());
    }

    assert(sets_flattened.size() == subsets_size_total);

    *set_beg_gpu = copy_to_device(set_beg);
    *sets_gpu = copy_to_device(sets_flattened);
}

GpuData::GpuData(const MarkShareFeas &ms_inst, const std::vector<std::vector<size_t>>& set1_subsets, const std::vector<size_t>& set1_scores, const std::vector<std::vector<size_t>>& set2_subsets, const std::vector<size_t>& set2_scores, const std::vector<std::vector<size_t>>& set3_subsets, const std::vector<size_t>& set3_scores, const std::vector<std::vector<size_t>>& set4_subsets, const std::vector<size_t>& set4_scores) : m_rows(ms_inst.m()), n_cols(ms_inst.n())
{
    this->matrix = copy_to_device(ms_inst.A());
    this->rhs = copy_to_device(ms_inst.b());

    copy_subsets(set1_subsets, &this->set1_subsets_beg, &this->set1_subsets);
    this->set1_scores = copy_to_device(set1_scores);
    copy_subsets(set2_subsets, &this->set2_subsets_beg, &this->set2_subsets);
    this->set2_scores = copy_to_device(set2_scores);
    copy_subsets(set3_subsets, &this->set3_subsets_beg, &this->set3_subsets);
    this->set3_scores = copy_to_device(set3_scores);
    copy_subsets(set4_subsets, &this->set4_subsets_beg, &this->set4_subsets);
    this->set4_scores = copy_to_device(set4_scores);
}

GpuData::~GpuData()
{
    hipFree(matrix);
    hipFree(rhs);
}

__global__ void check_sums(const size_t *val1, const size_t *val2, const size_t *rhs, size_t *solution, size_t n_val1, size_t n_val2, size_t m_rhs)
{
    int i1 = blockIdx.x * blockDim.x + threadIdx.x; // Thread for index i1
    int i2 = blockIdx.y * blockDim.y + threadIdx.y; // Thread for index i2

    if (i1 < n_val1 && i2 < n_val2)
    {
        bool feas = true;

        for (int j = 1; j < m_rhs; ++j)
        {
            const size_t sum = val1[i1 * m_rhs + j] + val2[i2 * m_rhs + j];
            if (sum != rhs[j])
            {
                feas = false;
                break;
            }
        }

        if (feas)
        {
            *solution = i1 * n_val2 + i2;
            return;
        }
    }
}

__global__ void binarySearchKeys(const __int128_t *sorted_keys, size_t num_sorted_keys,
                                 const __int128_t *query_keys, size_t *results, size_t num_query_keys)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_query_keys)
        return;

    __int128_t query_key = query_keys[idx];
    int64_t left = 0, right = num_sorted_keys - 1;

    // Binary search
    while (left <= right)
    {
        int64_t mid = left + (right - left) / 2;
        if (sorted_keys[mid] == query_key)
        {
            results[idx] = mid;
            return;
        }
        else if (sorted_keys[mid] < query_key)
        {
            left = mid + 1;
        }
        else
        {
            right = mid - 1;
        }
    }
}

__global__ void encodeVectors(const size_t *vectors, size_t num_vectors, size_t vector_size, __int128_t *encoded_keys)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_vectors)
        return;

    __int128_t key = 0;
    size_t base = 10000; // Maximum value per vector element
    for (size_t i = 0; i < vector_size; i++)
    {
        key = key * base + vectors[idx * vector_size + i];
    }
    encoded_keys[idx] = key;
}

std::pair<bool, std::pair<size_t, size_t>> evaluate_solutions_gpu(const GpuData &gpu_data, const std::vector<size_t> &scores_q1, const std::vector<size_t> &scores_q2, size_t n_q1, size_t n_q2)
{
    size_t result;
    size_t* d_rhs = gpu_data.rhs;
    size_t m = gpu_data.m_rows;

    size_t *d_scores_q1 = copy_to_device(scores_q1);
    size_t *d_scores_q2 = copy_to_device(scores_q2);

    size_t *d_solution;
    size_t sol_invalid = scores_q1.size() * scores_q1.size() + 1;
    hipMalloc(&d_solution, sizeof(size_t));
    hipMemcpy(d_solution, &sol_invalid, sizeof(size_t), hipMemcpyHostToDevice);

    // Define grid and block sizes
    dim3 blockDim(32, 32);                                                                    // Threads per block
    dim3 gridDim((n_q1 + blockDim.x - 1) / blockDim.x, (n_q2 + blockDim.y - 1) / blockDim.y); // Blocks per grid

    // Launch kernel
    check_sums<<<gridDim, blockDim>>>(d_scores_q1, d_scores_q2, d_rhs, d_solution, n_q1, n_q2, m);

    // Copy result back to host
    hipMemcpy(&result, d_solution, sizeof(size_t), hipMemcpyDeviceToHost);

    hipFree(d_scores_q1);
    hipFree(d_scores_q2);
    hipFree(d_rhs);
    hipFree(d_solution);

    hipDeviceSynchronize();
    if (result != sol_invalid)
    {
        printf("GPU found solution!\n");
        size_t i_q1 = result / n_q2;
        size_t i_q2 = result % n_q2;

        return {true, {i_q1, i_q2}};
    }
    else
    {
        return {false, {n_q1, n_q2}};
    }
}

__global__ void compute_required(const size_t* __restrict__ rhs, const size_t* __restrict__ scores_q1, size_t* __restrict__ required, size_t m, size_t n_q1)
{
    // Get the index of the current thread in the grid
    size_t i_q1 = blockIdx.x * blockDim.x + threadIdx.x;  // Corresponds to n_q1
    size_t i_m = threadIdx.y;                            // Corresponds to m

    // Check bounds
    if (i_q1 < n_q1 && i_m < m) {
        // Calculate the index
        size_t idx = i_q1 * m + i_m;
        // Perform the computation
        required[idx] = rhs[i_m] - scores_q1[idx];
    }
}

#define FULL_WARP_MASK    0xffffffff
#define WARP_SIZE 32

/* Simplicial factorization. */
static __forceinline__ __device__ int get_warp_id()
{
  int block_num_in_grid = blockIdx.x;
  assert(blockDim.x == warpSize);

  return block_num_in_grid;
}

static __forceinline__ __device__ unsigned get_lane_id()
{
  int thread_num_in_block = threadIdx.x;
  assert(blockDim.x == warpSize);
  assert(thread_num_in_block < warpSize);

  return thread_num_in_block;
}

/** Deterministically compute the sum values held by the threads in a warp; must be called by the whole warp. */
static __device__ size_t warp_sum_reduce(size_t value, /**< value's to compute the sum of */
  int thread                                  /**< thread id within the warp */
)
{
  assert(0 <= thread && thread < WARP_SIZE);
  /* Given a warp where each thread holds a (potentially different) value, compute the sum over all threads.
   * Say warpsize is 4 and v is v1,..v4 for each of the 4 threads. Then we have
   *
   * Lane       1      2      3      4
   *        [  v1,    v2,    v3,    v4  ]
   *                                        v += __shfl_down_sync(FULL_WARP_MASK, v, 2)
   *        [v1 + v3, v2 + v4,   x,   x ]
   *                                        v += __shfl_down_sync(FULL_WARP_MASK, v, 1)
   *        [v1 + v3 + v2 + v4, x, x, x ]
   *                                        __shfl_sync(FULL_WARP_MASK, v, 0)
   *        [ sum,   sum,   sum,   sum ]
   *
   * where sum = v1 + v3 + v2 + v4.
   *
   * __shfl_down_sync waits for all maked threads, then it retrieves for each lane the value at (lane + offset) %
   * width; width = WARP_SIZE here. __shfl_sync retrieves for each masked thread the value in the specified lane.
   */
  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
    value += __shfl_down_sync(FULL_WARP_MASK, value, offset);
  return __shfl_sync(FULL_WARP_MASK, value, 0);
}

static __device__ size_t warp_sum(const size_t* values,
    size_t n_values,
    const size_t* indices1,
    const size_t* indices2,
    size_t n_indices1,
    size_t n_indices2,
    size_t offset1,
    size_t offset2
)
{
  int lane_id = get_lane_id();
  size_t dot = 0;

  size_t iNz = lane_id;

  while (iNz < n_indices1 + n_indices2)
  {
    if (iNz < n_indices1)
    {
        assert(indices1[iNz] + offset1 < n_values);
        dot += values[indices1[iNz] + offset1];
    }
    else
    {
        assert(iNz - n_indices1 < n_indices2);
        assert(indices2[iNz - n_indices1] + offset2 < n_values);
        dot += values[indices2[iNz - n_indices1] + offset2];
    }

    iNz += WARP_SIZE;
  }

  dot = warp_sum_reduce(dot, lane_id);

  return dot;
}

static __device__ void warp_compute_scores_for_pair(const size_t* matrix, const size_t* indices1, const size_t* indices2, size_t n_indices1, size_t n_indices2, size_t* pair_scores, size_t m_rows, size_t n_cols, size_t offset1, size_t offset2)
{
    const size_t* matrix_ptr = matrix;

    for(size_t i_row = 0; i_row < m_rows; ++i_row)
    {
        pair_scores[i_row] = warp_sum(matrix_ptr, n_cols, indices1, indices2, n_indices1, n_indices2, offset1, offset2);

        matrix_ptr += n_cols;
    }
}

void combing_scores_gpu(const GpuData& gpu_data, std::vector<size_t>& scores, const std::vector<std::pair<size_t, size_t>>& pairs, const std::vector<size_t> offsets, bool first_sets)
{
    size_t* d_scores = copy_to_device(scores);
    size_t* d_pairs;
    hipMalloc(&d_pairs, pairs.size() * 2 * sizeof(size_t));
    hipMemcpy(d_pairs, pairs.data(), pairs.size() * 2 * sizeof(size_t), hipMemcpyHostToDevice);

    // size_t* d_first_subsets = first_sets ?  gpu_data.set1_subsets : gpu_data.set3_subsets;
    // size_t* d_first_subsets_beg = first_sets ?  gpu_data.set1_subsets_beg : gpu_data.set3_subsets_beg;
    // size_t* d_second_subsets = first_sets ?  gpu_data.set2_subsets : gpu_data.set4_subsets;
    // size_t* d_second_subsets_beg = first_sets ?  gpu_data.set2_subsets_beg : gpu_data.set4_subsets_beg;
    // size_t* d_pair_second_map = first_sets ? gpu_data.asc_indices_set2_weights : gpu_data.desc_indices_set4_weights;

    // const int n_pairs_per_warp = 500;
    // const int n_blocks = (pairs.size() + n_pairs_per_warp - 1) / n_pairs_per_warp;
    // size_t shared_mem_size = gpu_data.m_rows * gpu_data.n_cols * sizeof(size_t);

    // compute_scores_kernel<<<n_blocks, 32, shared_mem_size>>>(gpu_data.matrix, gpu_data.m_rows, gpu_data.n_cols, d_pairs, pairs.size(), d_first_subsets, d_first_subsets_beg, d_second_subsets, d_second_subsets_beg, d_pair_second_map, offsets[0], offsets[1], d_scores, n_pairs_per_warp);

    // hipMemcpy(scores.data(), d_scores, scores.size() * sizeof(size_t), hipMemcpyDeviceToHost);

    hipFree(d_scores);
    hipFree(d_pairs);
}

std::pair<bool, std::pair<size_t, size_t>> evaluate_solutions_gpu_hashing(const GpuData& gpu_data, const std::vector<size_t> &scores_q1, const std::vector<size_t> &scores_q2, size_t n_q1, size_t n_q2)
{
    size_t* d_rhs = gpu_data.rhs;
    size_t m_rows = gpu_data.m_rows;

    assert(m_rows > 0);

    auto profiler = std::make_unique<ScopedProfiler>("GPU hash setup");
    thrust::device_vector<size_t> d_required(scores_q1.size());
    thrust::device_vector<size_t> d_scores_q1(scores_q1);

    profiler = std::make_unique<ScopedProfiler>("GPU compute required");

    // Configure grid and block sizes
    dim3 blockDim(128, m_rows);  // 128 threads for i_q1, and each thread handles one value of m
    dim3 gridDim((n_q1 + blockDim.x - 1) / blockDim.x);
    compute_required<<<gridDim, blockDim>>>(d_rhs, thrust::raw_pointer_cast(d_scores_q1.data()), thrust::raw_pointer_cast(d_required.data()), m_rows, n_q1);

    profiler = std::make_unique<ScopedProfiler>("GPU data setup");

    thrust::device_vector<size_t> d_scores_q2(scores_q2);

    // THE ALGORITHM!
    // Allocate encoded key arrays on the GPU
    thrust::device_vector<__int128_t> d_keys1(n_q1);
    thrust::device_vector<__int128_t> d_keys2(n_q2);

    thrust::device_vector<size_t> d_indices(n_q1);
    thrust::sequence(d_indices.begin(), d_indices.end());

    profiler = std::make_unique<ScopedProfiler>("GPU encode");

    // Encode vectors into keys
    encodeVectors<<<(n_q1 + 255) / 256, 256>>>(thrust::raw_pointer_cast(d_required.data()), n_q1, m_rows, thrust::raw_pointer_cast(d_keys1.data()));
    encodeVectors<<<(n_q2 + 255) / 256, 256>>>(thrust::raw_pointer_cast(d_scores_q2.data()), n_q2, m_rows, thrust::raw_pointer_cast(d_keys2.data()));

    profiler = std::make_unique<ScopedProfiler>("GPU sort");

    // Sort the keys from l1
    thrust::sort_by_key(d_keys1.begin(), d_keys1.end(), d_indices.begin());

    profiler = std::make_unique<ScopedProfiler>("GPU search");

    thrust::device_vector<bool> d_result(n_q2);
    thrust::binary_search(thrust::device, d_keys1.begin(), d_keys1.end(), d_keys2.begin(), d_keys2.end(), d_result.begin());

    profiler = std::make_unique<ScopedProfiler>("Check results");
    thrust::host_vector<bool> result = d_result;

    for (size_t i_q2 = 0; i_q2 < n_q2; ++i_q2)
    {
        if (!result[i_q2])
            continue;

        thrust::host_vector<size_t> indices = d_indices;
        /* Retrieve i_q1. */
        auto iter = thrust::find(d_keys1.begin(), d_keys1.end(), d_keys2[i_q2]);
        profiler.reset();

        return {true, {indices[thrust::distance(d_keys1.begin(), iter)], i_q2}};
    }
    profiler.reset();

    return {false, {n_q1, n_q2}};
}